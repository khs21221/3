
#include <hip/hip_runtime.h>
extern "C" __global__ void 
madd(float* dst,  float* src1, float fac1, float* src2, float fac2, int N){
	int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
	if(i < N){
		dst[i] = fac1 * src1[i] + fac2 * src2[i];
	}
}

